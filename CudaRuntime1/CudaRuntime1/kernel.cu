#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <Windows.h>
#include <stdio.h>
#include <tchar.h>

__global__ void add(int a, int b, int* c) 
{
	*c = a + b;
}


int WINAPI WinMain(_In_ HINSTANCE hInstance, HINSTANCE hPrevInstance, LPSTR lpCmdLine, int nShowCmd)
{
	int c;
	int* dev_c;
	dev_c=new int(1);
	int handle=hipMalloc((void**)&dev_c, sizeof(int));
	add <<<2, 2>>> (2, 7, dev_c);
	hipMemcpy(&c,	dev_c, sizeof(int),	hipMemcpyDeviceToHost);
	char result[11];
	itoa(c, result, 10);
	MessageBox(NULL, TEXT(result), "result", MB_OK);
	hipFree(dev_c);

	size_t free_byte;
	size_t total_byte;

	hipError_t cuda_status = hipMemGetInfo(&free_byte, &total_byte);
	itoa(free_byte, result, 10);
	
	MessageBox(NULL, TEXT(result), "result", MB_OK);


	return 0;

}